#include "hip/hip_runtime.h"
#include "netWrapper.cuh"
#include "pipeline.h"
#include "gemmFactory.cuh"
#include <iostream>
#include <string>

PipelineBase::PipelineBase(vortexInitData* input_data, int nrank, int nranks, int vnranks)
	: input_data(input_data)
	, rank(nrank)
	, nranks(nranks)
	, vnranks(vnranks)
	, tmpBufferM(ptr_cast<cutlass::half_t>(input_data->tmp_buffer), input_data->tmp_buffer_size) {
	spdlog::info("rank: {}, nranks: {}", rank, nranks);
	std::string private_file_name = "log_" + std::to_string(rank) + ".txt";
	private_logger = spdlog::basic_logger_mt("private_logger" + std::to_string(rank), private_file_name, true);
	private_logger->set_level(spdlog::level::info);
	private_logger->log(spdlog::level::info, "rank: {}, nranks: {}", rank, nranks);
	private_logger->set_pattern("%v");
	spdlog::info("BaseFinished");
}



Pipeline::Pipeline(vortexInitData* input_data, int rank, int nranks, int vnranks, bool enable_offload)
	: PipelineBase(input_data, rank, nranks, vnranks)
	, enable_offload(enable_offload)
	, gemv_dep() {
	spdlog::info("Pipeline constructor");
	// sampled tokens 
	hipHostMalloc(&outputTokens, 4096*sizeof(int));

	if (enable_offload) {
		// should be equal to gemm batch size 
		// production = consumption
		// possible that one request need two cycles to offload 
		hipHostMalloc(&offloadKVCache, 1024*8192*2*sizeof(half)); // fix
		

		hipMalloc(&deviceOffloadKVCache, 1024*8192*2*sizeof(half));
		hipMalloc(&deviceLoadKVCache, 1024*8192*2*sizeof(half));

		
		int32_t * finished_idx_host = new int32_t[2048];
		int32_t * load_idx_host = new int32_t[2048];
		for	(int i = 0; i < 2048; i++) {
			finished_idx_host[i] = i;
			load_idx_host[i] = i;
		}

		hipMalloc(&finished_idx, 2048*sizeof(int32_t));
		hipMemcpy(finished_idx, finished_idx_host, 2048*sizeof(int32_t), hipMemcpyHostToDevice);

		hipMalloc(&load_idx, 2048*sizeof(int32_t));
		hipMemcpy(load_idx, load_idx_host, 2048*sizeof(int32_t), hipMemcpyHostToDevice);
		
	}

	spdlog::info("allocation done");
	init();
}

void Pipeline::StreamInit() {
	hipStreamCreate(&stream_gemm);
	hipStreamCreate(&stream_gemv);
	hipStreamCreate(&stream_net);
	hipStreamCreate(&stream_other);
	hipStreamCreate(&stream_cpy);
}

void Pipeline::setName() {
	SET_NAME_PTR(O1);
	SET_NAME_PTR(O2);
	SET_NAME_PTR(UG1);
	SET_NAME_PTR(UG2);
	SET_NAME_PTR(D1);
	SET_NAME_PTR(D2);
	SET_NAME_PTR(KQV1);
	SET_NAME_PTR(KQV2);
	SET_NAME_PTR(KQV3);
	SET_NAME_PTR(KQV4);
	SET_NAME_PTR(LOGITS1);
	SET_NAME_PTR(LOGITS2);

	SET_NAME_REF(AG_O1);
	SET_NAME_REF(AR_O2);
	SET_NAME_REF(AR_D1);
	SET_NAME_REF(AR1_D2);
	SET_NAME_REF(AR2_D2);
	SET_NAME_REF(AG1_GEMV);

	SET_NAME_REF(genEmbedding1);
	SET_NAME_REF(genEmbedding2_1);
	SET_NAME_REF(genEmbedding2_2);
	SET_NAME_REF(genEmbedding2_1_partial);
	SET_NAME_REF(genEmbedding2_2_partial);

	SET_NAME_REF(GEMV1);
	SET_NAME_REF(GEMV2);
	SET_NAME_REF(GEMV3);
	SET_NAME_REF(GEMV4);
	SET_NAME_REF(prefill);

	SET_NAME_REF(layerNormAttention1);
	SET_NAME_REF(layerNormAttention2_1);
	SET_NAME_REF(layerNormAttention2_2);
	SET_NAME_REF(layerNormFFN1);
	SET_NAME_REF(layerNormFFN2);
	SET_NAME_REF(layerNormModel1);

	SET_NAME_REF(activation1);
	SET_NAME_REF(activation2);

	for (int i = 0; i < 4; i++) {
		SET_NAME_REF(roPEAppends[i]);
	}

	SET_NAME_REF(pageAgg);
	SET_NAME_REF(pageDisp);
	SET_NAME_REF(splitTensor);

	SET_NAME_REF(maxSampler1);
	SET_NAME_REF(maxSampler2);

}


void Pipeline::GEMMOpInit() {

	cutlass::half_t beta(1);

	// Need to set the weight matrices of GEMM before initialize them. Otherwise there will be mysterious performance penalty.
	// Seems the GEMM->tensor_b's metadata will only be used during initialization and have no effect if specified later.
	setWeight(0);

	//TODO: fix it
	// KQV_START.init(buffer + offsetM.O1_IN,
	// 		weight_buffer + weight_offset,
	// 		buffer + offsetM.AG_KQV12 + residual_offset,
	// 		buffer + offsetM.O1_UG1,
	// 		KQV_START->M,
	// 		KQV_START->K,
	// 		KQV_START->M,
	// 		beta);

	O1->init(1);
	O2->init(0.125);
	UG1->init();
	UG2->init();
	D1->init(0.125);
	D2->init(0.125);
	KQV1->init();
	KQV2->init();
	KQV3->init();
	KQV4->init();

	LOGITS1->set_weight(input_data->weight.lm_head); // important
	LOGITS2->set_weight(input_data->weight.lm_head);

	LOGITS1->init();
	LOGITS2->init();

	LOGITS1->set_weight(input_data->weight.lm_head);
	LOGITS2->set_weight(input_data->weight.lm_head);

	for (auto gemm : gemms) {
		gemm->setStream(stream_gemm);
	}
	UG2->updateEventExistance(true, true);
}

void Pipeline::GEMVOpInit() {
	for (auto gemv : gemvs) {
		gemv->setStream(stream_gemv);
	}
	prefill.setStream(stream_gemv);

}


void Pipeline::OtherOpInit(){
	genEmbedding1.setWeight(input_data->weight.embedding);
	genEmbedding2_1_partial.setWeight(input_data->weight.embedding);
	genEmbedding2_1.setWeight(input_data->weight.embedding);
	genEmbedding2_2.setWeight(input_data->weight.embedding);
	genEmbedding2_2_partial.setWeight(input_data->weight.embedding);

	layerNormModel1.setWeight(input_data->weight.model_layernorm);
	// layerNormModel2_1.setWeight(input_data->weight.model_layernorm);
	// layerNormModel2_2.setWeight(input_data->weight.model_layernorm);

	genEmbedding1.setStream(stream_other);
	genEmbedding2_1_partial.setStream(stream_other);
	genEmbedding2_1.setStream(stream_other);
	genEmbedding2_2_partial.setStream(stream_other);
	genEmbedding2_2.setStream(stream_other);
	layerNormAttention1.setStream(stream_other);
	layerNormAttention2_1.setStream(stream_other);
	layerNormAttention2_2.setStream(stream_other);
	layerNormFFN1.setStream(stream_other);
	layerNormFFN2.setStream(stream_other);
	activation1.setStream(stream_gemm);
	activation2.setStream(stream_gemm);
	layerNormModel1.setStream(stream_other);
	// layerNormModel2_1.setStream(stream_other);
	// layerNormModel2_2.setStream(stream_other);
	for (int i = 0; i < 4; i++) {
		roPEAppends[i].setStream(stream_gemm);
		roPEAppends[i].updateEventExistance(false, true);
	}
	splitTensor.setStream(stream_net);
	roPEAppends[0].updateEventExistance(false, true);
	roPEAppends[3].updateEventExistance(false, true);
	KQV1->updateEventExistance(false, false);
	KQV2->updateEventExistance(false, true);
	KQV3->updateEventExistance(false, false);

	pageAgg.setStream(stream_cpy);
	pageDisp.setStream(stream_cpy);

	maxSampler1.setStream(stream_gemm);
	maxSampler2.setStream(stream_gemm);
}

void Pipeline::NetOpInit() {
	AG1_GEMV.setStream(stream_net);
	AR1_D2.setStream(stream_net);
	AR2_D2.setStream(stream_net);
	AG_O1.updateEventExistance(true, true);
	AG_O1.setStream(stream_net);
	AR_O2.setStream(stream_net);
	AR_O2.updateEventExistance(true, true);
	AR_D1.setStream(stream_net);
	AR1_D2.setEpsilon(1e-5);
	AR2_D2.setEpsilon(1e-5);
}

void PipelineBase::NetOpPrepare() {
#ifdef ENABLE_NETWORK
	// Initialize Communicator
	spdlog::info("Init communicator");
	std::shared_ptr<mscclpp::TcpBootstrap> bootstrap =
		std::make_shared<mscclpp::TcpBootstrap>(rank, nranks);
#ifdef ENABLE_MPI
	mscclpp::UniqueId uniqueId;
	if(rank == 0) uniqueId = bootstrap->createUniqueId();
	MPI_Bcast(&uniqueId, sizeof(uniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
	bootstrap->initialize(uniqueId);
#else
	if (rank == 0) shared_state.uniqueId = bootstrap->createUniqueId();
	worker_sync->barrier();
	bootstrap->initialize(shared_state.uniqueId);
#endif
	comm = std::make_shared<mscclpp::Communicator>(bootstrap);

	// Initialize Connections
	spdlog::info("Init connections");
	std::vector<mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>> connectionFutures;
	for(int r = 0; r < nranks; ++r) {
		if(r == rank) continue;
		mscclpp::Transport transport = mscclpp::Transport::CudaIpc;
		connectionFutures.push_back(comm->connectOnSetup(r, 0, transport));
	}
	comm->setup();
	std::transform(
		connectionFutures.begin(),
		connectionFutures.end(),
		std::back_inserter(connections),
		[](const mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>& future) {
			return future.get();
		});
	spdlog::info("Connections set up");

#ifdef ENABLE_MPI
	MPI_Barrier(MPI_COMM_WORLD);
#else
	worker_sync->barrier();
#endif

#endif

}

void Pipeline::init() {
	spdlog::info("Init stream");
	StreamInit();
	spdlog::info("Init pipeline");
#ifdef ENABLE_NETWORK
	spdlog::info("Init net");
	NetOpPrepare();
#endif
}


void Pipeline::ScheduleInit() {
	auto getMajor = [this](GEMM_NAME name, int x) {return getMajorType(config_data.gemm_op_tag[static_cast<int>(name)], x);};
	auto getDim = [this](GEMM_NAME name, int x) {return static_cast<PllmDimension>(getMajorType(config_data.gemm_op_tag[static_cast<int>(name)], x));};
	// Nanobatch metadata
	// KQV{1,2,3,4} output and AG{1,2}_GEMV both need to live on a contiguous buffer.
	// The KQV{1,2,3,4} => GEMV{1,2,3,4} dependency will be embedded in their kernel implementation
	const int kdim = MODEL_HEAD_DIM * MODEL_KV_HEADS;
	const int qdim = MODEL_HEAD_DIM * MODEL_QO_HEADS;
	const int vdim = MODEL_HEAD_DIM * MODEL_KV_HEADS;
	const auto& KQV_shared_output = tmpBufferM.allocTensor(config_data.global_batch_size, kdim + qdim + vdim, getMajor(GEMM_NAME::KQV1, 2));

	const auto& Q_shared = tmpBufferM.allocTensor(config_data.global_batch_size, qdim, getMajor(GEMM_NAME::KQV1, 2));
	
	gemvQ = tensor_cast<cutlass::half_t, half>(Q_shared);
	
	const auto& KQV_output_cutlass = KQV_shared_output.splitTensor(getDim(GEMM_NAME::KQV1, 2), KQV1->M, KQV2->M, KQV3->M, KQV4->M);
	KQV_output = tensor_cast<cutlass::half_t, half>(KQV_shared_output);

	// GEMV -> AG -> O1 -> UGD1
	// GEMV -> O2 (split K) -> AR -> UGD2
	const auto& GEMV_output_shared = tmpBufferM.allocTensor((O1->M + O2->M), MODEL_HIDDEN_DIM_PERGPU, getMajor(GEMM_NAME::O2, 0));
	spdlog::info(config_data.gemm_op_tag[static_cast<int>(GEMM_NAME::O1)]);
	spdlog::info(config_data.gemm_op_tag[static_cast<int>(GEMM_NAME::O2)]);
	gemvAggregateOutput = tensor_cast<cutlass::half_t, half>(GEMV_output_shared);

	const auto& GEMV_output_O1O2 = GEMV_output_shared.splitTensor(getDim(GEMM_NAME::O2, 0), O1->M, O2->M);
	const auto& AG1_GEMV_buffer = tmpBufferM.allocTensor(O1->M, O1->K, getMajor(GEMM_NAME::O1, 0));
	//spdlog::info("1");
	//spdlog::info("2");spdlog::info(static_cast<int>(getMajor(GEMM_NAME::O1, 1)));
	//spdlog::info("3");spdlog::info(static_cast<int>(getMajor(GEMM_NAME::O1, 2)));
	
	//// Nanoflow 1st batch
	// (prev)AG1_GEMV -> O1
	
	AG1_GEMV.init(comm, connections, rank, nranks, GEMV_output_O1O2[0], AG1_GEMV_buffer);
	//spdlog::info(static_cast<int>(AG1_GEMV.getOutput().layout));
	//spdlog::info(static_cast<int>(AG1_GEMV.getInput().layout));
	O1->setA(AG1_GEMV.getOutput());
	// TODO(gzuo): If I can decouple netwrapper init from setInput and setOutput, I can further simplify this.
	// O1 -> AG_O1
	
	O1->setD(tmpBufferM.allocTensor(O1->M, O1->N, getMajor(GEMM_NAME::O1, 2)));
	AG_O1.init(comm, connections, rank, nranks, O1->getD(), tmpBufferM.allocTensor(UG1->M, UG1->K, getMajor(GEMM_NAME::O1, 2)));
	// BEFORE_LN_FFN_1_TR.setInput(AG_O1.getOutput()).setOutput(tmpBufferM.allocTensor(UG1->M, UG1->K, getMajor(GEMM_NAME::D1, 2)));
	// AG_O1 -> LN_FFN1
	layerNormFFN1.setInput(AG_O1.getOutput()).setOutput(tmpBufferM.allocTensor(UG1->M, UG1->K, getMajor(GEMM_NAME::UG1, 0)));
	// LN_FFN1 -> UG1
	UG1->setA(layerNormFFN1.getOutput()).setOutput(tmpBufferM.allocTensor(UG1->M, UG1->N, getMajor(GEMM_NAME::UG1, 2)));
	// UG1 -> activation1
	activation1.setInput(UG1->getD()).setOutput(tmpBufferM.allocTensor(D1->M, D1->K, getMajor(GEMM_NAME::D1, 0)));
	// activation1 -> D1
	D1->setA(activation1.getOutput());
	// D1 -> AR_D1
	AR_D1.init(comm, connections, rank, nranks, tmpBufferM.allocTensor(D1->M, D1->N, getMajor(GEMM_NAME::D1, 2)), tmpBufferM.allocTensor(D1->M, D1->N, getMajor(GEMM_NAME::D1, 2)));
	D1->setOutput(AR_D1.getInput());
	D1->setC(AG_O1.getOutput()); // TODO same for D2
	// AR_D1 -> LN_Attention1
	layerNormAttention1.setInput(AR_D1.getOutput()).setOutput(tmpBufferM.allocTensor(D1->M, D1->N, getMajor(GEMM_NAME::D1, 2)));
	// LN_AT_1_TR.setInput(AR_D1.getOutput()).setOutput(tmpBufferM.allocTensor(D1->M, D1->N, getMajor(GEMM_NAME::O1, 2)));
	// The residual connection of the self-attention sublayer (before KQV12 and after O1)
	// TODO(gzuo): where is the residual connection of the MLP sublayer? Should add one before O1 and after D1?
	splitTensor.init(tensor_cast<cutlass::half_t, half>(AR_D1.getOutput()), tensor_cast<cutlass::half_t, half>(tmpBufferM.allocTensor(O1->M, O1->N, getMajor(GEMM_NAME::O1, 2))), nranks, rank);
	O1->setC(tensor_cast<half,cutlass::half_t>(splitTensor.output));
	// LN_Attention1 -> KQV1 | KQV2
	// NOTE: KQV1 and KQV2 share a contiguous output buffer because the following GEMV1 and GEMV2 might work with different batch sizes.
	const auto& KQV12_input = layerNormAttention1.getOutput().splitTensor(getDim(GEMM_NAME::KQV1, 0), KQV1->M, KQV2->M);
	KQV1->setA(KQV12_input[0]).setOutput(KQV_output_cutlass[0]);
	KQV2->setA(KQV12_input[1]).setOutput(KQV_output_cutlass[1]);

	// TODO: ropeAppend is not implemented yet

	// O1_TR.setInput(GEMV_output_O1O2[0]).setOutput(AG1_GEMV_buffer.getSubTensor(rank, vnranks, PllmDimension::ROW));


	//// Nanoflow 2nd batch
	// (prev)AG2_GEMV ->O2
	O2->setA(GEMV_output_O1O2[1]);
	// O2 -> AG_O2
	AR_O2.init(comm, connections, rank, nranks, tmpBufferM.allocTensor(UG2->M, UG2->K, getMajor(GEMM_NAME::O2, 2)), tmpBufferM.allocTensor(UG2->M, UG2->K, getMajor(GEMM_NAME::O2, 2)));
	O2->setD(AR_O2.getInput());
	// AG_O2 -> LN_FFN2
	layerNormFFN2.setInput(AR_O2.getOutput()).setOutput(tmpBufferM.allocTensor(UG2->M, UG2->K, getMajor(GEMM_NAME::UG2, 0)));
	// LN_FFN2 -> UG2
	UG2->setA(layerNormFFN2.getOutput()).setOutput(tmpBufferM.allocTensor(UG2->M, UG2->N, getMajor(GEMM_NAME::UG2, 2)));
	// UG2 -> activation2
	activation2.setInput(UG2->getD()).setOutput(tmpBufferM.allocTensor(D2->M, D2->K, getMajor(GEMM_NAME::D2, 0)));
	// activation2 -> D2
	D2->setA(activation2.getOutput()).setOutput(tmpBufferM.allocTensor(D2->M, D2->N, getMajor(GEMM_NAME::D2, 2)));
	D2->setC(AR_O2.getOutput());
	// D2 ->  D2_AR1 | D2_AR2
	const auto& AR12_D2_input = D2->getD().splitTensor(getDim(GEMM_NAME::D2, 2), KQV3->M, KQV4->M);
	const auto& shared_AR_output = tmpBufferM.allocTensor(KQV3->M + KQV4->M, KQV3->K, getMajor(GEMM_NAME::O2, 2));
	const auto& AR12_output = shared_AR_output.splitTensor(getDim(GEMM_NAME::O2, 2), KQV3->M, KQV4->M);

	const auto& AR12_before = tmpBufferM.allocTensor(O2->M, O2->N, getMajor(GEMM_NAME::O2, 2));
	AR12_before.clearContent();
	const auto& AR1_before_split = AR12_before.splitTensor(PllmDimension::ROW, KQV3->M, KQV4->M);

	AR1_D2.init(comm, connections, rank, nranks, AR12_D2_input[0], AR12_output[0], AR1_before_split[0]);
	AR2_D2.init(comm, connections, rank, nranks, AR12_D2_input[1], AR12_output[1], AR1_before_split[1]);
	// LN_Attention2_1/2 needs a shared contiguous buffer for the self-attention residual connection
	const auto& LN_Attn2_shared_output = tmpBufferM.allocTensor(KQV3->M + KQV4->M, KQV3->K, getMajor(GEMM_NAME::LOGITS2, 0));
	const auto& LN_Attn2_output = LN_Attn2_shared_output.splitTensor(getDim(GEMM_NAME::KQV1, 0), KQV3->M, KQV4->M);
	// TODO: O2 should setC(special column slice of LN_Attn2_shared_output)
	// e.g.,
	// const auto& O2_residule = tmpBufferM.allocSpan(O2->mn());
	// ExtractRankAsColumn.setInput(LN_Attn2_shared_output).setOutput(O2_residule);
	O2->setC(AR12_before);
	// D2_AR1 -> LN_Attention2_1 -> KQV3

	genEmbedding2_1.setOutput(tensor_cast<cutlass::half_t, half>(tmpBufferM.allocTensor(KQV3->M, KQV3->K, PllmLayout::ROW_MAJOR)));
	genEmbedding2_1_partial.setOutput(tensor_cast<cutlass::half_t, half>(AR1_before_split[0]
										.subtensor(AR1_before_split[0].dim1*rank/vnranks, AR1_before_split[0].dim1/vnranks)));
	layerNormAttention2_1.setInput(tensor_cast<half, cutlass::half_t>(genEmbedding2_1.getOutput())).setOutput(AR1_D2.getOutput());
	KQV3->setA(layerNormAttention2_1.getOutput());

	// D2_AR2 -> LN_Attention2_2 -> KQV4
	genEmbedding2_2.setOutput(tensor_cast<cutlass::half_t, half>(tmpBufferM.allocTensor(KQV4->M, KQV4->K, PllmLayout::ROW_MAJOR)));
	genEmbedding2_2_partial.setOutput(tensor_cast<cutlass::half_t, half>(AR1_before_split[1]
										.subtensor(AR1_before_split[1].dim1*rank/vnranks, AR1_before_split[1].dim1/vnranks)));
	layerNormAttention2_2.setInput(tensor_cast<half, cutlass::half_t>(genEmbedding2_2.getOutput())).setOutput(AR2_D2.getOutput());
	KQV4->setA(layerNormAttention2_2.getOutput());
	// KQV3 | KQV4 -> GEMV3 | GEMV4
	KQV3->setD(KQV_output_cutlass[2]);
	KQV4->setD(KQV_output_cutlass[3]);

	// connect setup output


	genEmbedding1.setOutput(tensor_cast<cutlass::half_t, half>(AR_D1.getOutput()));


	// connect logits generation

	layerNormModel1.setInput(AR_D1.getOutput()).setOutput(layerNormAttention1.getOutput());
	// layerNormModel2_1.setInput(AR1_D2.getOutput()).setOutput(layerNormAttention2_1.getOutput());
	// layerNormModel2_2.setInput(AR2_D2.getOutput()).setOutput(layerNormAttention2_2.getOutput());

	LOGITS1->setA(layerNormModel1.getOutput().getSubTensor(rank, vnranks, PllmDimension::ROW));
	LOGITS1->setOutput(tmpBufferM.allocTensor(LOGITS1->M, LOGITS1->N, getMajor(GEMM_NAME::LOGITS1, 2)));
	pllmTensor<int> sample_output1 = {(int*)tmpBufferM.alloc(LOGITS1->M * sizeof(int) / sizeof (half)), LOGITS1->M, 1, PllmLayout::ROW_MAJOR};
	maxSampler1.init(tensor_cast<cutlass::half_t, half>(LOGITS1->getD()), 
					 tensor_cast<cutlass::half_t, half>(tmpBufferM.allocTensor(LOGITS1->M, 1, PllmLayout::ROW_MAJOR)),
					 sample_output1); 

	LOGITS2->setA(shared_AR_output.getSubTensor(rank, vnranks, PllmDimension::ROW));
	LOGITS2->setOutput(tmpBufferM.allocTensor(LOGITS2->M, LOGITS2->N, getMajor(GEMM_NAME::LOGITS2, 2)));
	
	pllmTensor<int> sample_output2 = {(int*)tmpBufferM.alloc(LOGITS2->M * sizeof(int) / sizeof (half)), LOGITS2->M, 1, PllmLayout::ROW_MAJOR};
	maxSampler2.init(tensor_cast<cutlass::half_t, half>(LOGITS2->getD()), 
					 tensor_cast<cutlass::half_t, half>(tmpBufferM.allocTensor(LOGITS2->M, 1, PllmLayout::ROW_MAJOR)),
					 sample_output2);
}


void Pipeline::setWeight(int layer) {
	// Set weight before run the layer
	bool success = true;
	success &= O1->set_weight(input_data->weight.layer_weight[layer].W_O1);
	success &= O2->set_weight(input_data->weight.layer_weight[layer].W_O2);

	success &= UG1->set_weight(input_data->weight.layer_weight[layer].W_UG);
	success &= UG2->set_weight(input_data->weight.layer_weight[layer].W_UG);

	success &= D1->set_weight(input_data->weight.layer_weight[layer].W_D);
	success &= D2->set_weight(input_data->weight.layer_weight[layer].W_D);

	success &= KQV1->set_weight(input_data->weight.layer_weight[layer].W_KQV);
	success &= KQV2->set_weight(input_data->weight.layer_weight[layer].W_KQV);
	success &= KQV3->set_weight(input_data->weight.layer_weight[layer].W_KQV);
	success &= KQV4->set_weight(input_data->weight.layer_weight[layer].W_KQV);


	success &= layerNormAttention1.setWeight(input_data->weight.layer_weight[layer].W_LN_Attention);
	success &= layerNormAttention2_1.setWeight(input_data->weight.layer_weight[layer].W_LN_Attention);
	success &= layerNormAttention2_2.setWeight(input_data->weight.layer_weight[layer].W_LN_Attention);
	if (layer < MODEL_LAYER -1){
		success &= AR1_D2.setWeight(input_data->weight.layer_weight[layer+1].W_LN_Attention);
		success &= AR2_D2.setWeight(input_data->weight.layer_weight[layer+1].W_LN_Attention);
	} else{
		success &= AR1_D2.setWeight(input_data->weight.model_layernorm);
		success &= AR2_D2.setWeight(input_data->weight.model_layernorm);
	}


	success &= layerNormFFN1.setWeight(input_data->weight.layer_weight[layer].W_LN_FFN);
	success &= layerNormFFN2.setWeight(input_data->weight.layer_weight[layer].W_LN_FFN);

	if (!success) {
		spdlog::error("Failed to set weight for layer {}", layer);
	}
	if (layer == 0) {
		O2->set_beta(1);
	} else{
		O2->set_beta(1);
	}


	for (auto gemv : gemvs) {
		gemv->setKVData(input_data->kv_data[layer]);
	}
	prefill.setKVData(input_data->kv_data[layer]);
	roPEAppends[0].setKVData(input_data->kv_data[layer]);
	roPEAppends[1].setKVData(input_data->kv_data[layer]);
	roPEAppends[2].setKVData(input_data->kv_data[layer]);
	roPEAppends[3].setKVData(input_data->kv_data[layer]);	
}

double Pipeline::totalCompute() {
	double total = 0;
	for(auto gemm : gemms)
		total += gemm->totalCompute();
	return total;
}

void Pipeline::profileGEMM() {
	for(auto gemm : gemms)
		gemm->profile();
}

vortexOutputData Pipeline::run() {

	
	
	constexpr bool enableGraph = false;
	spdlog::info("Start run");
	setWeight(0);
	if(!enableGraph)
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_gemm));
	if(enableGraph) hipStreamBeginCapture(stream_gemm, hipStreamCaptureModeGlobal);
	hipEventRecord(events[EventManager::CAPTURE_GEMM_START], stream_gemm);
	hipStreamWaitEvent(stream_gemv, events[EventManager::CAPTURE_GEMM_START], 0);
	hipStreamWaitEvent(stream_net, events[EventManager::CAPTURE_GEMM_START], 0);
	constexpr int totalIter = 10000;

	// setup phase
	genEmbedding1.run().log(private_logger);
	genEmbedding2_1.run().log(private_logger);
	genEmbedding2_2.run().log(private_logger);
	genEmbedding2_1_partial.run().log(private_logger);
	genEmbedding2_2_partial.run().log(private_logger);


	splitTensor.run().log(private_logger);

	AR_D1.recordEndEvent();
	AR2_D2.recordEndEvent();
	AR1_D2.recordEndEvent();
	
	for(int iter = 0; iter < RUN_LAYER; ++iter) {
		private_logger->info(">>>>>>>>>>>>>>>>>>>>>>>>>> layer: {}", iter);
		// new starting point

		setWeight(iter%MODEL_LAYER);
							
		layerNormAttention1.wait(AR_D1).wait(genEmbedding1).run().log(private_logger);
		if (iter == 0){				
			layerNormAttention2_1.wait(genEmbedding2_1).run().log(private_logger);
			layerNormAttention2_2.wait(genEmbedding2_2).run().log(private_logger);
		}else{
			layerNormAttention2_1.wait(AR1_D2).skip();
			layerNormAttention2_2.wait(AR2_D2).skip();
		}

		OperatorWrapper* kqv_deps[]={&layerNormAttention1, &layerNormAttention1, &layerNormAttention2_1, &layerNormAttention2_2};
		
		for (int kqv_idx = 0; kqv_idx < 4; kqv_idx++) {
			std::string idx_str = std::to_string(kqv_idx);
			KQV_ptrs[kqv_idx]->wait(*kqv_deps[kqv_idx]).run().log(private_logger);
			if (roPEAppends[kqv_idx].dense_batch_size > 0)
				roPEAppends[kqv_idx].run().log(private_logger);
			else
				roPEAppends[kqv_idx].skip();
			
			if (gemvs[kqv_idx]->batch_size > 0)
				gemvs[kqv_idx]->wait(roPEAppends[kqv_idx]).run().log(private_logger);
			else
				gemvs[kqv_idx]->wait(roPEAppends[kqv_idx]).skip();
		}

		if (update_data.prefillNum > 0)
		{
			prefill.wait(roPEAppends[3]).run().log(private_logger);
			// gemv_dep.incCounter(gemv_dep.device_GEMV_ready, update_token_num - update_data.decodePrefillBorder, stream_gemv);
		}
			
		// int endpoint = update_token_num;
		// if (endpoint > config_data.nanobatch_1_size)
		// 	endpoint = config_data.nanobatch_1_size;

		// gemv_dep.blockUntilGEMVReady(stream_net, endpoint); // need fix
		// hipStreamWaitEvent(stream_net, events[EventManager::GEMV4_FINISH], 0);
		OperatorWrapper* AG_GEMV_dep;
		if (config_data.nanobatch_1_size == update_data.gemv_batch_size[0] + update_data.gemv_batch_size[1])
			AG_GEMV_dep = &GEMV2;
		else
			AG_GEMV_dep = &prefill;

		AG1_GEMV.setColumnwise().configRun(16, 1024, true).wait(AG_GEMV_dep).run().log(private_logger); // fix around
		

		O1->wait(splitTensor).wait(AG1_GEMV).run().log(private_logger); 
		
		AG_O1.setColumnwise().configRun(8, 1024, true).wait(O1).run().log(private_logger);

		layerNormFFN1.wait(AG_O1).run().log(private_logger);

		O2->wait_for_start(AG_O1).wait(GEMV4).wait(prefill).wait(KQV4).run().log(private_logger);

		AR_O2.configRun(8, 1024, true).wait(O2).run().log(private_logger);

		layerNormFFN2.wait(AR_O2).run().log(private_logger);


		// gemv_dep.clearAll(stream_net);

		UG1->wait_for_start(AR_O2).wait(layerNormFFN1).run().log(private_logger);

		activation1.run().log(private_logger);

		D1->run().log(private_logger);

		if (enable_offload) pageAgg.wait(activation1).run();



		UG2->wait(layerNormFFN2).run().log(private_logger);

		if (enable_offload) {
			hipStreamWaitEvent(stream_cpy, UG2->start_event);
			int split = 1;
			for (int j = 0; j < split; j++)
				hipMemcpyAsync(offloadKVCache+ j*2048*2*32*sizeof(half)*4/split, deviceOffloadKVCache, 2048*2*32*sizeof(half)*4/split, hipMemcpyDeviceToHost, stream_cpy);
			for (int j = 0; j < split; j++)
				hipMemcpyAsync(deviceLoadKVCache, offloadKVCache+ j*2048*2*32*sizeof(half) *4/split , 2048*2*32*sizeof(half)*4/split, hipMemcpyHostToDevice, stream_cpy);
		}

		activation2.run().log(private_logger);

		AR_D1.configRun(8, 1024, true).wait_for_start(*UG2).run().log(private_logger);

		splitTensor.run().log(private_logger);
		

		if (enable_offload) hipStreamWaitEvent(stream_cpy, AR_D1.end_event);
		if (enable_offload) pageDisp.run();

		D2->run().log(private_logger);
		
		AR1_D2.configRun(32, 1024, true).wait(D2).run().log(private_logger);

		AR2_D2.configRun(16, 1024, true).run().log(private_logger);
	}

	layerNormModel1.wait(AR_D1).run().log(private_logger);
	LOGITS1->wait(layerNormModel1).run().log(private_logger);
	maxSampler1.run().log(private_logger);

	LOGITS2->wait(AR1_D2).wait(AR2_D2).run().log(private_logger);
	maxSampler2.run().log(private_logger);

	hipMemcpyAsync(outputTokens, input_data->tmp_buffer, 2048*sizeof(int), hipMemcpyDeviceToHost, stream_cpy);
	hipEventRecord(events[EventManager::CAPTURE_GEMV_END], stream_gemv);
	hipEventRecord(events[EventManager::CAPTURE_NET_END], stream_net);
	hipStreamWaitEvent(stream_gemm, events[EventManager::CAPTURE_GEMV_END], 0);
	hipStreamWaitEvent(stream_gemm, events[EventManager::CAPTURE_NET_END], 0);

	// // End capture
	hipGraph_t graph;
	if(enableGraph) {
		hipStreamEndCapture(stream_gemm, &graph);
		if(graph == NULL) {
			spdlog::error("Failed to create graph");
			exit(1);
		}
		spdlog::info("Graph created");
		hipGraphExec_t instance;
		hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
		spdlog::info("Graph instantiated");
		CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_START], stream_gemm));
		for(int i = 0; i < 10; i ++)
			hipGraphLaunch(instance, stream_gemm);
		spdlog::info("Graph launched");
	}
	// Record an event when the GEMMs are complete
	CUDA_CHECK(hipEventRecord(events[EventManager::GEMM_TIMING_END], stream_gemm));

	// Wait for work on the device to complete.
	CUDA_CHECK(hipEventSynchronize(events[EventManager::GEMM_TIMING_END]));

	// Measure elapsed runtime
	float runtime_ms = 0;
	CUDA_CHECK(hipEventElapsedTime(&runtime_ms,
									events[EventManager::GEMM_TIMING_START],
									events[EventManager::GEMM_TIMING_END]));

	// Compute average runtime and GFLOPs.
	runtime_ms = double(runtime_ms);
	double gflops = totalCompute() / runtime_ms / 1e6;
	double bandwidth = sizeof(__half) * (96731136 + 1.25 / 80 * 160 * 1024 * 1024 * 1024 / 2) /
					   (runtime_ms / 1000) / (1 << 30);

	spdlog::info("Total running cost (ms) of one microbatch is {}", runtime_ms);

	// Copy output data back to host
	CUDA_CHECK(hipMemcpy(output_data.sampled_token_array1, maxSampler1.d_argMax.ptr, output_data.partial_num_1* sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(output_data.sampled_token_array2, maxSampler2.d_argMax.ptr, output_data.partial_num_2 * sizeof(int), hipMemcpyDeviceToHost));

	return output_data;
}

Pipeline::~Pipeline() {
	hipStreamDestroy(stream_gemm);
	hipStreamDestroy(stream_gemv);
	hipStreamDestroy(stream_net);
	hipStreamDestroy(stream_other);
}


void Pipeline::GEMVOpUpdate() {

	auto getDim = [this](GEMM_NAME name, int idx) {return static_cast<PllmDimension>(getMajorType(config_data.gemm_op_tag[static_cast<int>(name)], idx));};
	spdlog::info("Update GEMV");

	uint32_t arr[] = {update_data.gemv_batch_size[0], update_data.gemv_batch_size[1], update_data.gemv_batch_size[2], update_data.gemv_batch_size[3], update_data.prefillNum};
	std::span<uint32_t, 5> batch_sizes(arr, 5);
	std::span<int32_t, 4> gemv_num_blocks(update_data.gemv_num_blocks, 4);
	auto total_batch_size = std::accumulate(batch_sizes.begin(), batch_sizes.end(), 0);
	assert (total_batch_size == update_data.decodePrefillBorder + update_data.prefillNum);

	pllmTensor kv_indptr = pllmTensor{update_data.kv_indptr, total_batch_size + 1};
	const auto& kv_indptr_split = kv_indptr.splitTensor(PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);

	pllmTensor kv_last_page_len = pllmTensor{update_data.kv_last_page_len, total_batch_size};
	const auto& kv_last_page_len_split = kv_last_page_len.splitTensor(PllmDimension::ROW, batch_sizes);

	pllmTensor kv_indices = pllmTensor{update_data.kv_indices, MAX_PAGE_NUM};

	pllmTensor input_ptr = pllmTensor{update_data.input_indptr, total_batch_size + 1};
	const auto& input_ptr_split = input_ptr.splitTensor(PllmDimension::ROW, batch_sizes,/*overlap suffix*/ 1U);

	pllmTensor rev_input_indptr = pllmTensor{update_data.rev_input_indptr, config_data.global_batch_size};
	pllmTensor per_token_offset = pllmTensor{update_data.per_token_offset, config_data.global_batch_size};

	const auto& GEMV_input = gemvQ.splitTensor(getDim(GEMM_NAME::KQV1, 2),
									batch_sizes[0],
									batch_sizes[1],
									batch_sizes[2],
									batch_sizes[3],
									MAX_BATCH_SIZE - update_data.decodePrefillBorder);

	const auto& GEMV_output = gemvAggregateOutput.splitTensor(getDim(GEMM_NAME::KQV1, 2),
									batch_sizes[0],
									batch_sizes[1],
									batch_sizes[2],
									batch_sizes[3],
									MAX_BATCH_SIZE - update_data.decodePrefillBorder);

	const int kdim = MODEL_HEAD_DIM * MODEL_KV_HEADS;
	const int qdim = MODEL_HEAD_DIM * MODEL_QO_HEADS;
	const int vdim = MODEL_HEAD_DIM * MODEL_KV_HEADS;

	int req_num = update_data.decodePrefillBorder + update_data.prefillNum;
	
	int* host_input_ptr = new int[req_num + 1]; // each request start from ptr[i] and end before ptr[i+1]
	hipMemcpy(host_input_ptr, update_data.input_indptr, (req_num + 1) * sizeof(int), hipMemcpyDeviceToHost);
	update_token_num = host_input_ptr[req_num];
	const auto& Q_split = gemvQ.splitTensor(getDim(GEMM_NAME::KQV1, 2), KQV1->M, KQV2->M, KQV3->M, KQV4->M);
	const auto& KQV_split = KQV_output.splitTensor(getDim(GEMM_NAME::KQV1, 2), KQV1->M, KQV2->M, KQV3->M, KQV4->M);
	const auto& rev_input_indptr_split = rev_input_indptr.splitTensor(getDim(GEMM_NAME::KQV1, 2), KQV1->M, KQV2->M, KQV3->M, KQV4->M);
	const auto& per_token_offset_split = per_token_offset.splitTensor(getDim(GEMM_NAME::KQV1, 2), KQV1->M, KQV2->M, KQV3->M, KQV4->M);

	int token_remaining = update_token_num;
	for (int i = 0; i < 4; i++)
	{
		if (token_remaining > KQV_ptrs[i] -> M){
			roPEAppends[i].update(KQV_ptrs[i] -> M, KQV_split[i], Q_split[i], rev_input_indptr_split[i], per_token_offset_split[i], 
								kv_indices, kv_indptr, kv_last_page_len, gemv_dep.device_KQV_ready);
			token_remaining -= KQV_ptrs[i] -> M;
		}
		else{
			roPEAppends[i].update(token_remaining, KQV_split[i], Q_split[i], rev_input_indptr_split[i], per_token_offset_split[i], 
								kv_indices, kv_indptr, kv_last_page_len, gemv_dep.device_KQV_ready);
			token_remaining = 0;
		}
	}
	spdlog::info("RopeAppend Tokens: {}, {}, {}, {}", roPEAppends[0].dense_batch_size, roPEAppends[1].dense_batch_size, roPEAppends[2].dense_batch_size, roPEAppends[3].dense_batch_size);

	GEMV1.init(batch_sizes[0],
			gemv_num_blocks[0],
			kv_indptr_split[0],
			update_data.kv_indices,
			kv_last_page_len_split[0],
			GEMV_input[0],
			GEMV_output[0],
			gemv_dep.device_KQV_ready,
			gemv_dep.device_GEMV_ready);
	
	GEMV2.init(batch_sizes[1],
			gemv_num_blocks[1],
			kv_indptr_split[1],
			update_data.kv_indices,
			kv_last_page_len_split[1],
			GEMV_input[1],
			GEMV_output[1],
			gemv_dep.device_KQV_ready,
			gemv_dep.device_GEMV_ready);

	GEMV3.init(batch_sizes[2],
			gemv_num_blocks[2],
			kv_indptr_split[2],
			update_data.kv_indices,
			kv_last_page_len_split[2],
			GEMV_input[2],
			GEMV_output[2]);

	GEMV4.init(batch_sizes[3],
			gemv_num_blocks[3],
			kv_indptr_split[3],
			update_data.kv_indices,
			kv_last_page_len_split[3],
			GEMV_input[3],
			GEMV_output[3]);
	
	// ::log_tensor(spdlog::default_logger(), "input_ptr_split[4]", input_ptr_split[4], 1, 3);

	prefill.init(update_data.prefillNum,
				108,
				input_ptr_split[4],
				kv_indptr_split[4],
				update_data.kv_indices,
				kv_last_page_len_split[4],
				gemvQ,
				gemvAggregateOutput); 
				// use start address because prefill kernel get input use buffer[indptr[i]]

	// update page aggregation and page dispatch
	pageAgg.init(
		2,
		finished_idx,
		update_data.kv_indptr,
		update_data.kv_indices,
		deviceOffloadKVCache
	);
	pageAgg.setKVData(input_data->kv_data[0]);

	pageDisp.init(
		2,
		load_idx,
		update_data.kv_indptr,
		update_data.kv_indices,
		deviceLoadKVCache
	);
	pageDisp.setKVData(input_data->kv_data[0]);
	spdlog::info("finished GEMV update");	
}

void Pipeline::update(vortexUpdateData* update_data_) {


	this->update_data = *update_data_;

	// // debug 
	// spdlog::info("decode batch {}, prefill batch {}", update_data.decodePrefillBorder, update_data.prefillNum);
	// int total_batch = update_data.decodePrefillBorder + update_data.prefillNum;
	// int32_t * host_last_page_len = new int32_t[total_batch];
	// int32_t * host_indptr = new int32_t[total_batch + 1];
	// hipMemcpy(host_last_page_len, update_data.kv_last_page_len, total_batch * sizeof(int32_t), hipMemcpyDeviceToHost);
	// hipMemcpy(host_indptr, update_data.kv_indptr, (total_batch + 1) * sizeof(int32_t), hipMemcpyDeviceToHost);
	// int totalPage = host_indptr[total_batch];
	// int * host_indices = new int[totalPage];
	// hipMemcpy(host_indices, update_data.kv_indices, totalPage * sizeof(int), hipMemcpyDeviceToHost);
	// spdlog::info("indices: {}", formatCollection(host_indices, totalPage));
	// spdlog::info("last_page_len: {}", formatCollection(host_last_page_len, total_batch));
	// spdlog::info("indptr: {}", formatCollection(host_indptr, total_batch + 1));

	// int32_t * host_input_indptr = new int32_t[total_batch + 1];
	// hipMemcpy(host_input_indptr, update_data.input_indptr, (total_batch + 1) * sizeof(int32_t), hipMemcpyDeviceToHost);
	// spdlog::info("input_indptr: {}", formatCollection(host_input_indptr, total_batch + 1));
	
	// connect setup
	// split the tokens to 2 nano batch
	auto input_span = pllmTensor(update_data.input_tokens, config_data.global_batch_size, 1, PllmLayout::ROW_MAJOR);

	auto input_span1 = input_span.subtensor(0, config_data.nanobatch_1_size);

	auto input_span2_1 = input_span.subtensor(config_data.nanobatch_1_size, config_data.kqv3_size);
	auto input_span2_2 = input_span.subtensor(config_data.nanobatch_1_size + config_data.kqv3_size);

	auto partial_input_2_1 = input_span2_1.subtensor(input_span2_1.dim1 * rank / nranks, input_span2_1.dim1 / nranks);
	auto partial_input_2_2 = input_span2_2.subtensor(input_span2_2.dim1 * rank / nranks, input_span2_2.dim1 / nranks);

	

	genEmbedding1.setInput(input_span1);
	genEmbedding2_1.setInput(input_span2_1);
	genEmbedding2_2.setInput(input_span2_2);
	genEmbedding2_1_partial.setInput(partial_input_2_1);
	genEmbedding2_2_partial.setInput(partial_input_2_2);

	GEMVOpUpdate();

	// log first batch size
	spdlog::info("prefill: {}, decode: {}", update_data_->prefillNum, update_data_->decodePrefillBorder);
	spdlog::info("Batch size: {}, {}, {}, {}", update_data_->gemv_batch_size[0], update_data_->gemv_batch_size[1], update_data_->gemv_batch_size[2], update_data_->gemv_batch_size[3]);
}

void Pipeline::config(vortexConfigData* config_data){
	spdlog::info("Config pipeline");
	// // print all the members in config_data
	// spdlog::info("Config data: ");
	// spdlog::info("batch_size: {}", config_data->global_batch_size);
	// std::string s{};
	// for(auto i: config_data->gemm_op_tag) {
	// 	s += i + "   ";
	// }
	// spdlog::info("gemm_op_tag: {}", s);
	// spdlog::info("kqv_1: {}", config_data->kqv1_size);
	// spdlog::info("kqv_3: {}", config_data->kqv3_size);
	// spdlog::info("nanobatch: {}", config_data->nanobatch_1_size);

	this->config_data = * config_data;

	for(int i = 0; i < gemmNum; i++) {
		gemms[i] = generateGEMM(this->config_data.gemm_op_tag[i]);
		spdlog::info("GEMM {} created, tag: {}", i, this->config_data.gemm_op_tag[i]);
	}

	KQV_ptrs[0] = KQV1;
	KQV_ptrs[1] = KQV2;
	KQV_ptrs[2] = KQV3;
	KQV_ptrs[3] = KQV4;

	int globalbatch = config_data->global_batch_size;
	int nano1 = config_data->nanobatch_1_size;
	int nano2 = globalbatch - nano1;
	int kqv_batch[] = {config_data->kqv1_size, nano1 - config_data->kqv1_size, config_data->kqv3_size, nano2 - config_data->kqv3_size};

	O1 ->set_shape(nano1, MODEL_HIDDEN_DIM_PERGPU, MODEL_HIDDEN_DIM);
	O2 ->set_shape(nano2, MODEL_HIDDEN_DIM, MODEL_HIDDEN_DIM_PERGPU);
	UG1 ->set_shape(nano1, UG_N, MODEL_HIDDEN_DIM);
	UG2 ->set_shape(nano2, UG_N, MODEL_HIDDEN_DIM);
	D1 ->set_shape(nano1, MODEL_HIDDEN_DIM, MODEL_FF_DIM_GPU);
	D2 ->set_shape(nano2, MODEL_HIDDEN_DIM, MODEL_FF_DIM_GPU);
	KQV1 -> set_shape(kqv_batch[0], KQV_N, MODEL_HIDDEN_DIM);
	KQV2 -> set_shape(kqv_batch[1], KQV_N, MODEL_HIDDEN_DIM);
	KQV3 -> set_shape(kqv_batch[2], KQV_N, MODEL_HIDDEN_DIM);
	KQV4 -> set_shape(kqv_batch[3], KQV_N, MODEL_HIDDEN_DIM);
	LOGITS1 -> set_shape(nano1/nranks, 32000, MODEL_HIDDEN_DIM);
	LOGITS2 -> set_shape(nano2/nranks, 32000, MODEL_HIDDEN_DIM);


	activation1.config(nano1, UG_N/2);
	activation2.config(nano2, UG_N/2);

	setName();

	spdlog::info("Init schedule");
	ScheduleInit();
	spdlog::info("Init gemm");
	GEMMOpInit();
	// No longer need GEMVOpInit and OtherOpInit at this moment.
	spdlog::info("Init gemv");
	GEMVOpInit();
	spdlog::info("Init other");
	OtherOpInit();

	NetOpInit();

	// init the output 
	output_data = vortexOutputData();
	output_data.partial_num_1 = config_data->nanobatch_1_size / nranks;
	output_data.partial_num_2 = (config_data->global_batch_size - config_data->nanobatch_1_size)/ nranks;
	output_data.sampled_token_array1 = new int[config_data->nanobatch_1_size];
	output_data.sampled_token_array2 = new int[config_data->global_batch_size - config_data->nanobatch_1_size];
	output_data.global_batch_size = config_data->global_batch_size;

	D2->updateEventExistance(true, true);
}