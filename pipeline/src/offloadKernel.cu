#include "hip/hip_runtime.h"
#include "offloadKernel.cuh"
#include <cstdio>

__device__ void pageCopy(half* input, half* output){

    int copyIter = PAGE_MEM_SIZE / sizeof(float4) / blockDim.x;
    // printf("copyIter: %d\n", copyIter);
    float4* input4 = (float4*)input;
    float4* output4 = (float4*)output;

    for (int i = 0; i < copyIter; i++){
        output4[i * blockDim.x + threadIdx.x] = input4[i * blockDim.x + threadIdx.x];
    }
}

__global__ void moveKVcacheKernel(int finished_req_num, int32_t * finished_index, 
                                       int32_t* kv_indptr, int32_t* kv_indices, half* host_ptr, half* kv_data, bool host_to_gpu){
    for (int i = 0; i < finished_req_num; i++){
        int idx = finished_index[i];
        int start = kv_indptr[idx];
        int end = kv_indptr[idx + 1];

        for (int j = start + blockIdx.x; j < end; j += gridDim.x){
            int page_idx = kv_indices[j];
            half* page = kv_data + page_idx * PAGE_MEM_SIZE;
            half* host_page = host_ptr + j * PAGE_MEM_SIZE;
            // printf("page_idx: %d\n", page_idx);
            if (host_to_gpu)
                pageCopy(host_page, page);
            else
                pageCopy(page, host_page);
        }
    }
}

